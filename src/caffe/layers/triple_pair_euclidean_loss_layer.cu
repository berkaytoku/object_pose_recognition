#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void TriplePairEuclideanLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  vector<double> temp0, temp, pair;
  float m = 0.01;
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      xixj_diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      xixj_diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());  // (a_i-b_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      xixj_dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  Dtype margin = this->layer_param_.contrastive_loss_param().margin();
  bool legacy_version =
      this->layer_param_.contrastive_loss_param().legacy_version();
  Dtype loss(0.0);
  
  double tempDenominator = 0.0;
  
  for (int i = 0; i < bottom[0]->num(); ++i) {
      tempDenominator += diff_sq_.cpu_data()[i];
  }

  tempDenominator = sqrt(tempDenominator) + m;

  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[2]->gpu_data(),  // b
      xixk_diff_.mutable_gpu_data());  // a_i-b_i

  caffe_gpu_powx(
      count,
      xixk_diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      xixk_dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  
  double tempNumerator = 0.0;
  for(int j=0; j<bottom[0]->num(); j++) {
	tempNumerator += diff_sq_.mutable_cpu_data()[j];  	
  }

  tempNumerator = sqrt(tempNumerator);
  Dtype dist = std::max(1-(tempNumerator/tempDenominator), 0.0);
  loss += dist;
  
  caffe_gpu_sub(
      count,
      bottom[3]->gpu_data(),  // a
      bottom[4]->gpu_data(),  // b
      xixj_p_diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      xixj_p_dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  double denomForPair = 0.0;
  for(int k=0; k<bottom[0]->num(); k++) {
	denomForPair += diff_sq_.mutable_cpu_data()[k];  	
  }
  loss += denomForPair;
  
  //loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels, int bottom_index, const Dtype *x,
    Dtype *bottom_diff, const Dtype *xixj_dist_sq_, const Dtype *xixk_dist_sq_) {
  Dtype bottom_diff_val(0.0);
  CUDA_KERNEL_LOOP(i, count) {
    //int n = i / channels;  // the num index, to access y and dist_sq
	 
		  if(bottom_index < 3){ //triple
				//gradient of loss equation
				if(bottom_index == 0){ //dLoss/dxi
					bottom_diff_val = sqrt(xixk_dist_sq_[i])/ sqrt(xixj_dist_sq_[i]);
				}
				else if (bottom_index == 1){ //dLoss/dxj
					bottom_diff_val = -(x[i] / sqrt(xixj_dist_sq_[i]));
				}			
				else if (bottom_index == 2){ //dLoss/dxk
					bottom_diff_val = -(sqrt(xixk_dist_sq_[i]) / x[i]);
				}
		  }
		  else if(bottom_index >= 3){ //pair
			  //gradient of loss equation
			  if (bottom_index == 3){ //dLoss/dxi_p
				  bottom_diff_val = 2 * x[i];
			  }
			  else if (bottom_index == 4){ //dLoss/dxj_p
				  bottom_diff_val = -(2 * x[i]);
			  }  
		  }
		  if (bottom_diff_val > 0.0){
			bottom_diff[i] = bottom_diff_val;
		  }
		  else{
			bottom_diff[i] = 0;
		  }
				
  }
}

template <typename Dtype>
void TriplePairEuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	for(int i = 0; i < 5; ++i){
	  Dtype* bout = bottom[i]->mutable_cpu_diff();
	  //int num = bottom[i]->num();
	  int count = bottom[i]->count();
	  int channels = bottom[i]->channels();
	   if (propagate_down[i]) {
		  CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
		  count, channels, i, bottom[i]->gpu_data(), bottom[i]->mutable_gpu_diff(), xixj_dist_sq_.gpu_data(), xixk_dist_sq_.gpu_data());
		  CUDA_POST_KERNEL_CHECK;
	  }
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(TriplePairEuclideanLossLayer);

}  // namespace caffe
