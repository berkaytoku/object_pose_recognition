#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void TriplePairEuclideanLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  vector<double> temp0, temp, pair;
  float m = 0.01;

  /*
  for (int j=0; j < 10; j++) {
      LOG(INFO) << *(bottom[0]->cpu_data() + j);
  }
  */

  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      xixj_diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      xixj_diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());  // (a_i-b_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      xixj_dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  Dtype margin = this->layer_param_.contrastive_loss_param().margin();
  bool legacy_version =
      this->layer_param_.contrastive_loss_param().legacy_version();
  Dtype loss(0.0);
  
  double tempDenominator = 0.0;
  
  for (int i = 0; i < bottom[0]->num(); ++i) {
      tempDenominator += diff_sq_.cpu_data()[i];
  }

  tempDenominator = sqrt(tempDenominator) + m;

  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[2]->gpu_data(),  // b
      xixk_diff_.mutable_gpu_data());  // a_i-b_i

  caffe_gpu_powx(
      count,
      xixk_diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      xixk_dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  
  double tempNumerator = 0.0;
  for(int j=0; j<bottom[0]->num(); j++) {
	tempNumerator += diff_sq_.mutable_cpu_data()[j];  	
  }

  tempNumerator = sqrt(tempNumerator);
  Dtype dist = std::max(1-(tempNumerator/tempDenominator), 0.0);
  loss += dist;
  
  caffe_gpu_sub(
      count,
      bottom[3]->gpu_data(),  // a
      bottom[4]->gpu_data(),  // b
      xixj_p_diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      xixj_p_dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  double denomForPair = 0.0;
  for(int k=0; k<bottom[0]->num(); k++) {
	denomForPair += diff_sq_.mutable_cpu_data()[k];  	
  }
  loss += denomForPair;
  //printf("Loss = %f \n", loss);
  //loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels, int bottom_index,
    Dtype *bottom_diff, const Dtype *xixj_diff_, const Dtype *xixk_diff_, const Dtype *xixj_p_diff_, const Dtype *xixj_dist_sq_, const Dtype *xixk_dist_sq_) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
		if(bottom_index < 3){ //triple
			//derivative of max function
			if(sqrt(xixk_dist_sq_[n]) / (sqrt(xixj_dist_sq_[n]) + Dtype(1e-2)) < 1){
				//gradient of loss equation
				if(bottom_index == 0){ //dLoss/dxi
					bottom_diff[i] = -((xixk_diff_[i]/sqrt(xixk_dist_sq_[n] + Dtype(1e-2))) * (sqrt(xixj_dist_sq_[n]) + Dtype(1e-2)) - (sqrt(xixk_dist_sq_[n]) * (xixj_diff_[i] / (xixj_dist_sq_[n] + Dtype(1e-2)))));
					bottom_diff[i] /= powf(sqrt(xixj_dist_sq_[n]) + Dtype(1e-2), 2);
					//printf("dLoss/dxi = %f \n", bottom_diff[i]);
				}
				else if (bottom_index == 1){ //dLoss/dxj
					bottom_diff[i] = sqrt(xixk_dist_sq_[n]) * (xixj_diff_[i] / sqrt(xixj_dist_sq_[n] + Dtype(1e-2)));
					bottom_diff[i] /= powf(sqrt(xixj_dist_sq_[n]) + Dtype(1e-2), 2);
				}			
				else if (bottom_index == 2){ //dLoss/dxk
					bottom_diff[i] = xixk_diff_[i] / sqrt(xixk_dist_sq_[n] + Dtype(1e-2)) ;
					bottom_diff[i] /= sqrt(xixj_dist_sq_[n]) + Dtype(1e-2);
				}
			}
			else{
				bottom_diff[i] = 0;
			}
		}
		else if(bottom_index >= 3){ //pair 
			//gradient of loss equation
			if (bottom_index == 3){ //dLoss/dxi_p
				bottom_diff[i] = 2 * xixj_p_diff_[i];
			}
			else if (bottom_index == 4){ //dLoss/dxj_p
				bottom_diff[i] = -(2 * xixj_p_diff_[i]);
			}  
		}
				
  }
}

template <typename Dtype>
void TriplePairEuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	for(int i = 0; i < 5; ++i){
	  Dtype* bout = bottom[i]->mutable_cpu_diff();
	  //int num = bottom[i]->num();
	  int count = bottom[i]->count();
	  int channels = bottom[i]->channels();
	   if (propagate_down[i]) {
		  CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
		  count, channels, i, bottom[i]->mutable_gpu_diff(), 
		  xixj_diff_.gpu_data(), xixk_diff_.gpu_data(), xixj_p_diff_.gpu_data(),
		  xixj_dist_sq_.gpu_data(), xixk_dist_sq_.gpu_data());
		  CUDA_POST_KERNEL_CHECK;
	  }
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(TriplePairEuclideanLossLayer);

}  // namespace caffe
