#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void TriplePairEuclideanLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();

  /*
  for (int j=0; j < 10; j++) {
      LOG(INFO) << *(bottom[0]->cpu_data() + j);
  }
  */

  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      xixj_diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      xixj_diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());  // (a_i-b_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      xixj_dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2

    caffe_gpu_sub(
        count,
        bottom[0]->gpu_data(),  // a
        bottom[2]->gpu_data(),  // b
        xixk_diff_.mutable_gpu_data());  // a_i-b_i

    caffe_gpu_powx(
        count,
        xixk_diff_.mutable_gpu_data(),  // a_i-b_i
        Dtype(2),
        diff_sq_.mutable_gpu_data());

    caffe_gpu_gemv(
        CblasNoTrans,
        bottom[0]->num(),
        bottom[0]->channels(),
        Dtype(1.0),
        diff_sq_.gpu_data(),  // (a_i-b_i)^2
        summer_vec_.gpu_data(),
        Dtype(0.0),
        xixk_dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2

    caffe_gpu_sub(
        count,
        bottom[3]->gpu_data(),  // a
        bottom[4]->gpu_data(),  // b
        xixj_p_diff_.mutable_gpu_data());  // a_i-b_i
    caffe_gpu_powx(
        count,
        xixj_p_diff_.mutable_gpu_data(),  // a_i-b_i
        Dtype(2),
        diff_sq_.mutable_gpu_data());
    caffe_gpu_gemv(
        CblasNoTrans,
        bottom[0]->num(),
        bottom[0]->channels(),
        Dtype(1.0),
        diff_sq_.gpu_data(),  // (a_i-b_i)^2
        summer_vec_.gpu_data(),
        Dtype(0.0),
        xixj_p_dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2

  Dtype loss(0.0);
  for (int i = 0; i < bottom[0]->num(); ++i) {
        loss += std::max(Dtype(0.0), static_cast<Dtype>(1-(sqrt(xixk_dist_sq_.cpu_data()[i])/(sqrt(xixj_dist_sq_.cpu_data()[i]) + Dtype(1e-2)))));
        loss += xixj_p_dist_sq_.cpu_data()[i];
  }


    top[0]->mutable_cpu_data()[0] = loss / static_cast<Dtype>(bottom[0]->num());
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels, int bottom_index,
    Dtype *bottom_diff, const Dtype *xixj_diff_, const Dtype *xixk_diff_, const Dtype *xixj_p_diff_, const Dtype *xixj_dist_sq_, const Dtype *xixk_dist_sq_, const Dtype alpha) {
  CUDA_KERNEL_LOOP(i, count) {
	//printf("bottom_index = %f \n", bottom_index);
	//printf("channel = %f \n", channels);
    
	int n = i / channels;  // the num index, to access y and dist_sq
		if(bottom_index < 3){ //triple
			//derivative of max function
			//printf("Loss = %f \n", 1-(sqrt(xixk_dist_sq_[n]) / (sqrt(xixj_dist_sq_[n]) + Dtype(1e-2))));
			//LOG(INFO) << "Loss : " << 1-(sqrt(xixk_dist_sq_[n]) / (sqrt(xixj_dist_sq_[n]) + Dtype(1e-2)));
			if(sqrt(xixk_dist_sq_[n]) / (sqrt(xixj_dist_sq_[n]) + Dtype(1e-2)) < 1){
				//gradient of loss equation
				if(bottom_index == 0){ //dLoss/dxi
					bottom_diff[i] = -((xixk_diff_[i]/(sqrt(xixk_dist_sq_[n]) + Dtype(1e-3))) * (sqrt(xixj_dist_sq_[n]) + Dtype(1e-2)) - (sqrt(xixk_dist_sq_[n]) * (xixj_diff_[i] / (sqrt(xixj_dist_sq_[n]) + Dtype(1e-3)))));
					bottom_diff[i] /= powf(sqrt(xixj_dist_sq_[n]) + Dtype(1e-2), 2);
					//printf("dLoss/dxi = %f \n", bottom_diff[i]);
				}
				else if (bottom_index == 1){ //dLoss/dxj
					bottom_diff[i] = -(sqrt(xixk_dist_sq_[n]) * (xixj_diff_[i] / (sqrt(xixj_dist_sq_[n]) + Dtype(1e-3))));
					bottom_diff[i] /= powf(sqrt(xixj_dist_sq_[n]) + Dtype(1e-2), 2);
				}			
				else if (bottom_index == 2){ //dLoss/dxk
					bottom_diff[i] = xixk_diff_[i] / (sqrt(xixk_dist_sq_[n]) + Dtype(1e-3)) ;
					bottom_diff[i] /= sqrt(xixj_dist_sq_[n]) + Dtype(1e-2);
				}
			}
			else{
				bottom_diff[i] = 0;
			}
		}
		else if(bottom_index >= 3){ //pair 
			//gradient of loss equation
			if (bottom_index == 3){ //dLoss/dxi_p
				bottom_diff[i] = 2 * xixj_p_diff_[i];
			}
			else if (bottom_index == 4){ //dLoss/dxj_p
				bottom_diff[i] = -(2 * xixj_p_diff_[i]);
			}  
		}
bottom_diff[i] *= alpha;
  }
}

template <typename Dtype>
void TriplePairEuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	for(int i = 0; i < 5; ++i){
	   if (propagate_down[i]) {
            int count = bottom[0]->count();
            int channels = bottom[0]->channels();
          const Dtype alpha = top[0]->cpu_diff()[0] / static_cast<Dtype>(bottom[0]->num());
		  CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
		  count, channels, i, bottom[i]->mutable_gpu_diff(), 
		  xixj_diff_.gpu_data(), xixk_diff_.gpu_data(), xixj_p_diff_.gpu_data(),
		  xixj_dist_sq_.gpu_data(), xixk_dist_sq_.gpu_data(), alpha);
		  CUDA_POST_KERNEL_CHECK;
		  //LOG(INFO) << channels<< "   bindex: " << i;
	  }
	  
	  
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(TriplePairEuclideanLossLayer);

}  // namespace caffe
